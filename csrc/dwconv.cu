#include "hip/hip_runtime.h"
#include "dwconv.h"
#include "utils.h"
#include <ATen/cuda/HIPContext.h>

template <typename Dtype>
__device__ void block_reduce(Dtype* data) {
  int tid = threadIdx.x;
#pragma unroll
  for (int i = kCudaThreadsNum / 2; i > 0; i /= 2) {
    data[tid] += data[tid + i];
    __syncthreads();
  }
}

template <typename Dtype>
__global__ void dwconv_forward_backward_kernel(
    Dtype* out, const Dtype* __restrict__ data,
    const Dtype* __restrict__ weight, const int64_t* __restrict__ neigh,
    const int64_t height, const int64_t channel, const int64_t kngh,
    const int64_t nthreads) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    int64_t h = i / channel;
    int64_t c = i % channel;
    Dtype value = Dtype(0);
    for (int k = 0; k < kngh; ++k) {
      int64_t ni = neigh[h * kngh + k];
      if (ni >= 0) {
        value += weight[k * channel + c] * data[ni * channel + c];
      }
    }
    out[i] = value;
  }
}

template <typename Dtype>
__global__ void dwconv_weight_backward_kernel(
    Dtype* out, const Dtype* __restrict__ grad, const Dtype* __restrict__ data,
    const int64_t* __restrict__ ineigh, const int64_t height, const int64_t channel,
    const int64_t kngh, const int64_t nthreads) {
  __shared__ Dtype weights[kCudaThreadsNum];
  CUDA_KERNEL_LOOP(i, nthreads) {
    int64_t h = i / channel;
    int64_t c = i % channel;
    for (int k = 0; k < kngh; ++k) {
      int64_t ni = ineigh[h * kngh + k];
      int64_t tid = threadIdx.x;
      weights[tid] = ni >= 0 ? data[ni * channel + c] * grad[i] : Dtype(0);
      __syncthreads();
      block_reduce(weights);
      if (tid == 0) {
        int n = i / kCudaThreadsNum;
        out[(n * kngh + k) * channel + c] = weights[0];
      }
    }
  }
}

__global__ void inverse_neigh_kernel(
    int64_t* ineigh, const int64_t* __restrict__ neigh, const int64_t height, 
    const int64_t kngh, const int64_t nthreads) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    int64_t h = i / kngh;
    int64_t c = i % kngh;
    int64_t j = neigh[i];
    if (j >= 0) {
      ineigh[j * kngh + c] = h;
    }
  }
}

Tensor inverse_neigh(Tensor neigh) {
  int64_t height = neigh.size(0);
  int64_t kngh = neigh.size(1);
  int64_t nthreads = height * kngh;
  Tensor ineigh = torch::full_like(neigh, -1);
  auto stream = at::cuda::getCurrentCUDAStream();
  inverse_neigh_kernel
      <<<CudaGetBlocks(nthreads), kCudaThreadsNum, 0, stream>>>(
      ineigh.data_ptr<int64_t>(), neigh.data_ptr<int64_t>(), height, kngh, nthreads);
  return ineigh;
}

Tensor dwconv_forward_backward(Tensor data, Tensor weight, Tensor neigh) {
  // data: (N, C), weight: (K, 1, C), neigh: (N, K)
  int64_t height = data.size(0);
  int64_t channel = data.size(1);
  int64_t nthreads = height * channel;
  int64_t kngh = neigh.size(1);
  Tensor out = torch::zeros_like(data);
  auto stream = at::cuda::getCurrentCUDAStream();
  dwconv_forward_backward_kernel<float>
      <<<CudaGetBlocks(nthreads), kCudaThreadsNum, 0, stream>>>(
      out.data_ptr<float>(), data.data_ptr<float>(), weight.data_ptr<float>(),
      neigh.data_ptr<int64_t>(), height, channel, kngh, nthreads);
  CUDA_POST_KERNEL_CHECK;
  return out;
}

Tensor dwconv_weight_backward(Tensor grad, Tensor data, Tensor neigh) {
  int64_t height = data.size(0);
  int64_t channel = data.size(1);
  int64_t nthreads = height * channel;
  int64_t kngh = neigh.size(1);
  Tensor out = grad.new_zeros({height / kCudaThreadsNum, kngh, 1, channel});
  auto stream = at::cuda::getCurrentCUDAStream();
  dwconv_weight_backward_kernel<float>
      <<<CudaGetBlocks(nthreads), kCudaThreadsNum, 0, stream>>>(
      out.data_ptr<float>(), grad.data_ptr<float>(), data.data_ptr<float>(),
      neigh.data_ptr<int64_t>(), height, channel, kngh, nthreads);
  CUDA_POST_KERNEL_CHECK;
  return out.sum(0);
}
